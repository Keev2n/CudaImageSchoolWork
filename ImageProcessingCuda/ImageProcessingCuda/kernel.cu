#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "ImageProcessing.h"
#include <math.h>
#include <iostream>


using namespace std;

__global__ void ImageToGrayScale_CUDA(unsigned char* Image, int Row, int Col, int Channels, unsigned char* Image2);

void image_toGrayScale_Cuda(unsigned char* Image, int Row, int Col, int Channels, unsigned char* Image2) {
	unsigned char* dev_Image = NULL;
	unsigned char* dev_Image2 = NULL;

	hipMalloc((void**)&dev_Image, Row * Col * Channels);
	hipMalloc((void**)&dev_Image2, Row * Col);

	hipMemcpy(dev_Image, Image, Row * Col * Channels, hipMemcpyHostToDevice);
	hipMemcpy(dev_Image2, Image2, Row * Col, hipMemcpyHostToDevice);

	int threadNumber = 16;

	int blockX = (Col / threadNumber) + 1;
	int blockY = (Row / threadNumber) + 1;

	ImageToGrayScale_CUDA <<< dim3(blockX,blockY), dim3(threadNumber, threadNumber) >> > (dev_Image, Row, Col, Channels, dev_Image2);
	cout << hipGetLastError() << endl;

	hipMemcpy(Image, dev_Image, Row * Col * Channels, hipMemcpyDeviceToHost);
	hipMemcpy(Image2, dev_Image2, Row * Col, hipMemcpyDeviceToHost);

	hipFree(Image);
	hipFree(Image2);
}

__global__ void ImageToGrayScale_CUDA(unsigned char* Image, int Row, int Col, int Channels, unsigned char* Image2) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	if (x < Col && y < Row) {
		int grayOffset = y * Col + x;
		int rgbOffset = grayOffset * Channels;

		unsigned char b = Image[rgbOffset];
		unsigned char g = Image[rgbOffset + 1]; 
		unsigned char r = Image[rgbOffset + 2];


		Image2[grayOffset] = 0.21f * r + 0.71f * g + 0.07f * b;
	}
}